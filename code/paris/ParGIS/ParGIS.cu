#include "hip/hip_runtime.h"
#include <float.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "ParGIS.h"
__constant__ float sax_breakpointingpu[] ={-2.10003938216135,-2.06623181534436,-2.03283385699339,-1.99984225624759,-1.96725376224613,-1.93506512412816,-1.90327309103284,-1.87187441209932,-1.84086583646678,-1.81024411327436,-1.78000599166123,-1.75014822076655,-1.72066754972946,-1.69156072768914,-1.66282450378474,-1.63445562715541,-1.60645084694032,-1.57880691227863,-1.55152057230949,-1.52458857617206,-1.49800767300551,-1.47177461194898,-1.44588614214165,-1.42033901272266,-1.39512997283118,-1.37025577160636,-1.34571315818736,-1.32149888171335,-1.29760969132347,-1.27404233615690,-1.25079356535278,-1.22786012805028,-1.20523877338855,-1.18292625050675,-1.16091930854405,-1.13921469663960,-1.11780916393255,-1.09669945956208,-1.07588233266733,-1.05535453238746,-1.03511280786164,-1.01515390822902,-0.995474582628758,-0.976071580200017,-0.956941650081954,-0.938081541413728,-0.919488003334496,-0.901157784983417,-0.883087635499651,-0.865274304022356,-0.847714539690689,-0.830405091643811,-0.813342709020879,-0.796524140961053,-0.779946136603490,-0.763605445087349,-0.747498815551789,-0.731622997135969,-0.715974738979046,-0.700550790220181,-0.685347899998530,-0.670362817453254,-0.655592291723510,-0.641033071948457,-0.626681907267254,-0.612535546819059,-0.598590739743031,-0.584844235178328,-0.571292782264110,-0.557933130139534,-0.544762027943759,-0.531776224815945,-0.518972469895249,-0.506347512320829,-0.493898101231846,-0.481620985767457,-0.469512915066820,-0.457570638269096,-0.445790904513441,-0.434170462939015,-0.422706062684976,-0.411394452890483,-0.400232382694694,-0.389216601236769,-0.378343857655865,-0.367610901091141,-0.357014480681756,-0.346551345566869,-0.336218244885638,-0.326011927777221,-0.315929143380778,-0.305966640835466,-0.296121169280445,-0.286389477854873,-0.276768315697909,-0.267254431948711,-0.257844575746437,-0.248535496230247,-0.239323942539300,-0.230206663812752,-0.221180409189764,-0.212241927809494,-0.203387968811101,-0.194615281333742,-0.185920614516577,-0.177300717498764,-0.168752339419462,-0.160272229417829,-0.151857136633025,-0.143503810204207,-0.135208999270534,-0.126969452971165,-0.118781920445259,-0.110643150831974,-0.102549893270468,-0.0944988968999002,-0.0864869108594294,-0.0785106842882140,-0.0705669663254127,-0.0626525061101840,-0.0547640527816864,-0.0468983554790787,-0.0390521633415194,-0.0312222255081671,-0.0234052911181804,-0.0155981093107179,-0.00779742922493818,0.00,0.00779742922493841,0.0155981093107181,0.0234052911181806,0.0312222255081673,0.0390521633415196,0.0468983554790789,0.0547640527816867,0.0626525061101842,0.0705669663254129,0.0785106842882143,0.0864869108594296,0.0944988968999004,0.102549893270468,0.110643150831974,0.118781920445259,0.126969452971166,0.135208999270535,0.143503810204207,0.151857136633025,0.160272229417830,0.168752339419462,0.177300717498764,0.185920614516577,0.194615281333742,0.203387968811101,0.212241927809494,0.221180409189765,0.230206663812752,0.239323942539300,0.248535496230247,0.257844575746437,0.267254431948711,0.276768315697909,0.286389477854873,0.296121169280445,0.305966640835466,0.315929143380778,0.326011927777221,0.336218244885638,0.346551345566869,0.357014480681756,0.367610901091141,0.378343857655865,0.389216601236769,0.400232382694694,0.411394452890483,0.422706062684976,0.434170462939015,0.445790904513441,0.457570638269096,0.469512915066821,0.481620985767457,0.493898101231846,0.506347512320829,0.518972469895249,0.531776224815945,0.544762027943760,0.557933130139534,0.571292782264110,0.584844235178328,0.598590739743031,0.612535546819059,0.626681907267254,0.641033071948457,0.655592291723510,0.670362817453254,0.685347899998530,0.700550790220181,0.715974738979046,0.731622997135969,0.747498815551789,0.763605445087349,0.779946136603489,0.796524140961053,0.813342709020879,0.830405091643811,0.847714539690689,0.865274304022356,0.883087635499651,0.901157784983417,0.919488003334496,0.938081541413728,0.956941650081954,0.976071580200017,0.995474582628758,1.01515390822902,1.03511280786164,1.05535453238746,1.07588233266733,1.09669945956208,1.11780916393255,1.13921469663960,1.16091930854405,1.18292625050675,1.20523877338855,1.22786012805028,1.25079356535278,1.27404233615690,1.29760969132347,1.32149888171335,1.34571315818736,1.37025577160636,1.39512997283118,1.42033901272266,1.44588614214165,1.47177461194898,1.49800767300551,1.52458857617206,1.55152057230949,1.57880691227863,1.60645084694032,1.63445562715541,1.66282450378474,1.69156072768914,1.72066754972946,1.75014822076655,1.78000599166123,1.81024411327436,1.84086583646678,1.87187441209932,1.90327309103284,1.93506512412816,1.96725376224613,1.99984225624759,2.03283385699339,2.06623181534436,2.10003938216135};

__global__ void calculate_lbdold(const sax_type * const saxarray,const float * const paa, const long int M, const int N,float * const sax_breakpoints,bool * positionarray,const float BSF,float segmentsize) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float distance = 0;


	int i=0;
        		float breakpoint_lower = 0; // <-- TODO: calculate breakpoints.
        		float breakpoint_upper = 0; // <-- - || -


	for (int j = thid; j < M; j += gridDim.x*blockDim.x)
	{
		distance=0;
		for (i=0; i<N; i++) 
		{
			sax_type v = saxarray[j*N+i];
			sax_type region_lower = v;//shift operation 
			sax_type region_upper = (~((int)MAXFLOAT) | region_lower);
			if (region_lower == 0)
			{
				breakpoint_lower = -2000000;
			}
			else
			{
				breakpoint_lower = sax_breakpointingpu[region_lower];//(float)(region_lower-128)*(region_lower-128)/16484.0f;//sax_breakpoints[region_lower];
			}

			if (region_upper == 256 - 1) 
			{
				breakpoint_upper = +2000000;
			}
			else
			{
				breakpoint_upper = sax_breakpointingpu[region_lower+1];//(float)(region_upper+1-128)*(region_upper+1-128)/16484.0f;//sax_breakpoints[region_upper+1];//search in a list(why?)
			}


			if (breakpoint_lower > paa[i]) 
			{
				distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
			}
			else if(breakpoint_upper < paa[i])
			{
				distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
			}

		}

		if(segmentsize*distance<BSF)
		{positionarray[j]=true;}
		else
		{positionarray[j]=false;}
	}
}

__global__ void calculate_lbd(const sax_type * const saxarray,const float * const paa, const long int M, const int N,bool * positionarray,const float BSF,float segmentsize) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float distance = 0;
	float lbsf=BSF/segmentsize;

	int i=0;
        		float breakpoint_lower = 0; // <-- TODO: calculate breakpoints.
        		float breakpoint_upper = 0; // <-- - || -

	
	for (int j = thid; j < M; j += gridDim.x*blockDim.x)
	{
		distance=0;
		for (i=0; i<N; i++) {
                	if(distance<lbsf)
		{
        	
        		sax_type v = saxarray[j*N+i];

        		sax_type region_lower = v ;//shift operation 
        		sax_type region_upper = (~((int)MAXFLOAT) | region_lower);



        	
        		if (region_lower == 0)
			{
	            		breakpoint_lower = -2000000;
				float breaku=((float)region_lower-127.0f)/128.0f;
            			breakpoint_upper = breaku*(1.1362582192*breaku*breaku+0.99800);//sax_breakpoints[region_upper+1];//search in a list(why?)
				if(breakpoint_upper < paa[i])
				{
            				distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
        			}
			}
        		else if (region_upper == 256 - 1) 
			{
            			breakpoint_upper = +2000000;
				float breakx=((float)region_lower-128.0f)/128.0f;
           			breakpoint_lower = breakx*(breakx*breakx*1.1362582192+0.99800);//sax_breakpoints[region_lower];
				if (breakpoint_lower > paa[i]) 
				{
            				distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
        			}
        		}
        		else 
			{
				float breakx=((float)region_lower-128.0f)/128.0f;
           			breakpoint_lower = breakx*(breakx*breakx*1.1362582192+0.99800);//sax_breakpoints[region_lower];
				if (breakpoint_lower > paa[i]) 
				{
            				distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
        			}
				else
				{
					float breaku=((float)region_lower-127.0f)/128.0f;
            				breakpoint_upper = breaku*(1.1362582192*breaku*breaku+0.99800);//sax_breakpoints[region_upper+1];//search in a list(why?)
					if(breakpoint_upper < paa[i])
					{
            					distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
        				}
        			} 

        		}
		}
						        		
    	}

		if(distance<lbsf)
		{positionarray[j]=true;}
		else
		{positionarray[j]=false;}
	}
}



__global__ void calculate_lbdfloat(const sax_type * const saxarray,const float * const paa, const long int M, const int N,float * positionarray,const float BSF, float segmentsize) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float distance = 0;
	

	int i=0;
        		float breakpoint_lower = 0; // <-- TODO: calculate breakpoints.
        		float breakpoint_upper = 0; // <-- - || -


	for (int j = thid; j < M; j += gridDim.x*blockDim.x)
	{
		distance=0;
		for (i=0; i<N; i++) 
		{
			if(segmentsize*distance<BSF)
			{
				sax_type v = saxarray[j*N+i];
				sax_type region_lower = v ;//shift operation 
				sax_type region_upper = (~((int)MAXFLOAT) | region_lower);
				if (region_lower == 0)
				{
					breakpoint_lower = -2000000;
					float breaku=((float)region_lower-127.0f)/128.0f;
					breakpoint_upper = breaku*(1.1362582192*breaku*breaku+0.99800);//sax_breakpoints[region_upper+1];//search in a list(why?)
					if(breakpoint_upper < paa[i])
					{
						distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
					}
				}
				else if (region_upper == 256 - 1)
				{
					breakpoint_upper = +2000000;
					float breakx=((float)region_lower-128.0f)/128.0f;
					breakpoint_lower = breakx*(breakx*breakx*1.1362582192+0.99800);//sax_breakpoints[region_lower];
					if (breakpoint_lower > paa[i]) 
					{
						distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
					}
				}
				else
				{
					float breakx=((float)region_lower-128.0f)/128.0f;
					breakpoint_lower = breakx*(breakx*breakx*1.1362582192+0.99800);//sax_breakpoints[region_lower];
					if (breakpoint_lower > paa[i]) 
					{
						distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
					}
					else
					{
						float breaku=((float)region_lower-127.0f)/128.0f;
						breakpoint_upper = breaku*(1.1362582192*breaku*breaku+0.99800);//sax_breakpoints[region_upper+1];//search in a list(why?)
						if(breakpoint_upper < paa[i])
						{
							distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
						}
					}
				}
			}
		}
		positionarray[j]=segmentsize*distance;
	}
}


__global__ void calculate_lbdfloattable(const sax_type * const saxarray,const float * const paa, const long int M, const int N,float * positionarray,const float BSF, float segmentsize) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float distance = 0;
	

	int i=0;
        		float breakpoint_lower = 0; // <-- TODO: calculate breakpoints.
        		float breakpoint_upper = 0; // <-- - || -


	for (int j = thid; j < M; j += gridDim.x*blockDim.x)
	{
		distance=0;
		for (i=0; i<N; i++) 
		{
			if(segmentsize*distance<BSF)
			{
				sax_type v = saxarray[j*N+i];
				sax_type region_lower = v ;//shift operation 
				sax_type region_upper = (~((int)MAXFLOAT) | region_lower);
				if (region_lower == 0)
				{
					breakpoint_lower = -2000000;
					breakpoint_upper = sax_breakpointingpu[region_upper+1];//search in a list(why?)
					if(breakpoint_upper < paa[i])
					{
						distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
					}
				}
				else if (region_upper == 256 - 1)
				{
					breakpoint_upper = +2000000;
					breakpoint_lower = sax_breakpointingpu[region_lower];
					if (breakpoint_lower > paa[i]) 
					{
						distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
					}
				}
				else
				{
					breakpoint_lower = sax_breakpointingpu[region_lower];
					if (breakpoint_lower > paa[i]) 
					{
						distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
					}
					else
					{
						breakpoint_upper = sax_breakpointingpu[region_upper+1];//search in a list(why?)
						if(breakpoint_upper < paa[i])
						{
							distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
						}
					}
				}
			}
		}
		positionarray[j]=segmentsize*distance;
	}
}






extern "C" float* initialGPU(float *qts, float *gqts, sax_type *saxarray, sax_type *gsaxarray, float *dictionary, float *gdictionary,unsigned long datasize,float *sax_breakpoints )
{

	hipSetDevice(0); 
	hipMalloc(&gqts, sizeof(float)*16); 
	return gqts;
}
extern "C" float* initialgqts(float *gqts)
{
		hipMalloc(&gqts, sizeof(float)*16); 
		return gqts;
}
extern "C" void GPUsyn()
{
hipDeviceSynchronize();
}




extern "C" float* initialgdictionary(float *gdictionary)
{
		hipMalloc(&gdictionary, sizeof(float)*FULLSIZE);

		return gdictionary;
}
extern "C" bool* initialgposbitmap(bool *gposbitmap,unsigned long datasize)
{
	hipMalloc(&gposbitmap, sizeof(bool)*datasize); 
		return gposbitmap;
}
extern "C" bool* initialposbitmap(bool *posbitmap,unsigned long datasize)
{
	hipHostMalloc(&posbitmap, sizeof(bool)*datasize); 
		return posbitmap;
}
extern "C" float* initialgposbitmapfloat(float *gposbitmap,unsigned long datasize)
{
	hipMalloc(&gposbitmap, sizeof(float)*datasize); 
		return gposbitmap;
}
extern "C" float* initialposbitmapfloat(float *posbitmap,unsigned long datasize)
{
hipHostMalloc(&posbitmap, sizeof(float)*datasize); 
		return posbitmap;
}

extern "C" sax_type* initialgsaxarray(sax_type *gsaxarray,unsigned long datasize)
{
	hipMalloc(&gsaxarray, sizeof(sax_type)*datasize*16); 

		return gsaxarray;
}
extern "C" sax_type* initialsaxarray(sax_type *saxarray,unsigned long datasize)
{
	hipHostMalloc(&saxarray, sizeof(sax_type)*datasize*16); 

		return saxarray;
}
extern "C" void initialdevice()
{
	hipSetDevice(0);
}
extern "C" void gpumemcpy(sax_type *gsaxarray,sax_type *saxarray,unsigned long datasize)
{	

	hipMemcpy(gsaxarray, saxarray,sizeof(sax_type)*datasize*16,hipMemcpyHostToDevice);
}
extern "C" void gpusaxgridmemcpy(sax_type *gsaxarray,sax_type *saxarray,unsigned long datasize)
{	
	hipMemcpy(gsaxarray, saxarray,sizeof(sax_type)*datasize*16,hipMemcpyHostToDevice);
}

extern "C" void gpudictionarymemcpy(float *gdictionary,float *sax_breakpoints)
{	
int offset = ((256 - 1) * (256 - 2)) / 2;
	hipMemcpy(gdictionary, &sax_breakpoints[offset-1], sizeof(float)*FULLSIZE,hipMemcpyHostToDevice);
	hipDeviceSynchronize();
}





extern "C" void GPUfree(void *devicememorypointer)
{
	hipFree(devicememorypointer);
}






extern "C" void LBDfloatstreamGPU(sax_type *saxarray, float *posbitmap,float * qts,float * gqts,float BSF,long unsigned datasize,float * gposbitmap,int segmentnumber,float segmentsize)
{
	int streamnumber=10;
	hipMemcpy(gqts, qts,sizeof(float)*segmentnumber,hipMemcpyHostToDevice);
	hipStream_t streams[streamnumber];
	for(int i=0;i<streamnumber;i++)
	{
		hipStreamCreate(&streams[i]);
	}

	for(int i=0;i<streamnumber;i++)
	{
		calculate_lbdfloattable<<<200,500,10,streams[i]>>> (saxarray+i*datasize*segmentnumber/streamnumber,gqts, datasize/streamnumber, segmentnumber, gposbitmap+i*datasize/streamnumber,BSF,segmentsize); 
        hipMemcpyAsync(posbitmap+i*datasize/streamnumber, gposbitmap+i*datasize/streamnumber, sizeof(float)*datasize/streamnumber,hipMemcpyDeviceToHost,streams[i]);
	}
	hipDeviceSynchronize();
}




extern "C" void LBDstreamGPU(sax_type *saxarray, bool *posbitmap,float * qts,float * gqts,float BSF,long unsigned datasize,bool * gposbitmap,int segmentnumber,float segmentsize)
{
	int streamnumber=20;
	hipMemcpy(gqts, qts,sizeof(float)*segmentnumber,hipMemcpyHostToDevice);
	hipStream_t streams[streamnumber];
	for(int i=0;i<streamnumber;i++)
	{
		hipStreamCreate(&streams[i]);
	}
	for(int i=0;i<streamnumber;i++)
	{
		calculate_lbd<<<200,500,10,streams[i]>>> (saxarray+i*datasize*segmentnumber/streamnumber,gqts, datasize/streamnumber, segmentnumber,gposbitmap+i*datasize/streamnumber,BSF, segmentsize); 
		hipMemcpyAsync(posbitmap+i*datasize/streamnumber, gposbitmap+i*datasize/streamnumber, sizeof(bool)*datasize/streamnumber,hipMemcpyDeviceToHost,streams[i]);
	}
	hipDeviceSynchronize();
}


extern "C" void SIMSlowertableGPU(sax_type *saxarray, bool *posbitmap,float * qts,float * gqts,float BSF,long unsigned datasize,bool * gposbitmap,int segmentnumber,float segmentsize,float *gdictionary)
{
	int streamnumber=20;
	hipMemcpy(gqts, qts,sizeof(float)*segmentnumber,hipMemcpyHostToDevice);
	hipStream_t streams[streamnumber];
	for(int i=0;i<streamnumber;i++)
	{
		hipStreamCreate(&streams[i]);
	}
	for(int i=0;i<streamnumber;i++)
	{
		//calculate_lbd<<<200,500,10,streams[i]>>> (saxarray+i*datasize*segmentnumber/streamnumber,gqts, datasize/streamnumber, segmentnumber,gposbitmap+i*datasize/streamnumber,BSF, segmentsize); 
		calculate_lbdold<<<200,500,10,streams[i]>>>(saxarray+i*datasize*segmentnumber/streamnumber,gqts, datasize/streamnumber, segmentnumber,gdictionary,gposbitmap+i*datasize/streamnumber,BSF, segmentsize);
		hipMemcpyAsync(posbitmap+i*datasize/streamnumber, gposbitmap+i*datasize/streamnumber, sizeof(bool)*datasize/streamnumber,hipMemcpyDeviceToHost,streams[i]);
	}
	hipDeviceSynchronize();
}

















/*
extern "C" void SIMSlowerGPU(sax_type *saxarray, bool *posbitmap,float * qts,float * gqts,float BSF,long unsigned datasize,bool * gposbitmap,float *gdictionary)
{

	hipMemcpy(gqts, qts,sizeof(float)*16,hipMemcpyHostToDevice);
	calculate_lbd<<<200,200,10>>> (saxarray,gqts, datasize, 16, gposbitmap,BSF,16); 

	hipMemcpy(posbitmap, gposbitmap, sizeof(bool)*datasize,hipMemcpyDeviceToHost);
//hipDeviceSynchronize();


}
extern "C" void SIMSlowerGPUfloat(sax_type *saxarray, float *posbitmap,float * qts,float * gqts,float BSF,long unsigned datasize,float * gposbitmap,float *gdictionary)
{

	hipMemcpy(gqts, qts,sizeof(float)*16,hipMemcpyHostToDevice);
	calculate_lbdfloat<<<200,500,10>>> (saxarray,gqts, datasize, 16,gdictionary, gposbitmap,BSF,16); 
	hipMemcpy(posbitmap, gposbitmap, sizeof(float)*datasize,hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
}

extern "C" void initialGPU_grid2(float *qts, float *gqts, sax_type **saxarray, sax_type **gsaxarray, bool **posbitmap, bool **gposbitmap, float *dictionary, float *gdictionary, unsigned long datasize,unsigned long *gridsize,float *sax_breakpoints )
{
	hipSetDevice(0); 

	for(int i =0;i<65536;i++)
	{
		hipHostMalloc(&saxarray[i], sizeof(sax_type)*gridsize[i]*16); 
		hipMalloc(&gposbitmap[i], sizeof(float)*gridsize[i]); 
		hipMalloc(&gsaxarray[i], sizeof(float)*gridsize[i]*16); 
		hipHostMalloc(&posbitmap[i], sizeof(float)*gridsize[i]); 
		hipMemcpy(gsaxarray[i], saxarray[i],sizeof(float)*gridsize[i]*16,hipMemcpyHostToDevice);
	}
	hipHostMalloc(&dictionary, 257*sizeof(float)); 
	hipHostMalloc(&qts, sizeof(float)*256); 
	int offset = ((256 - 1) * (256 - 2)) / 2;
	//memcpy(dictionary,&sax_breakpoints[offset-1],sizeof(float)*257);

	hipMalloc(&gdictionary, sizeof(float)*257);
	hipMalloc(&gqts, sizeof(float)*256); 

	hipMemcpy(gdictionary, &sax_breakpoints[offset-1], sizeof(float)*257,hipMemcpyHostToDevice);
}
extern "C" void SIMSlowerGPUgridstream(sax_type **saxarray, bool **posbitmap,float * qts,float * gqts,float BSF,long unsigned datasize,bool ** gposbitmap,float *gdictionary,long int *nodemap, unsigned long *gridnumber)
{
	int streamnumber=(int)datasize;
	hipStream_t streams[streamnumber];
	hipMemcpy(gqts, qts,sizeof(float)*16,hipMemcpyHostToDevice);
	for(int i=0;i<streamnumber;i++)
	{
		hipStreamCreate(&streams[i]);
	}

	for(long int i=0;i<streamnumber;i++)
	{

		calculate_lbd2<<<500,100,10,streams[i]>>> (saxarray[nodemap[i]],gqts, gridnumber[nodemap[i]], 16,gdictionary, gposbitmap[nodemap[i]],BSF); 
        hipMemcpyAsync(posbitmap[nodemap[i]], gposbitmap[nodemap[i]], sizeof(bool)*gridnumber[nodemap[i]],hipMemcpyDeviceToHost,streams[i]);
	}
	for(long int i=0;i<datasize;i++)
	{
		//hipMemcpy(posbitmap[nodemap[i]], gposbitmap[nodemap[i]], sizeof(bool)*gridnumber[nodemap[i]],hipMemcpyDeviceToHost,streams[i]);
	}
	hipDeviceSynchronize();
}
extern "C" void SIMSstreamlowerGPU2(sax_type *saxarray, bool *posbitmap,float * qts,float * gqts,float BSF,unsigned long int startnumber,unsigned long int stopnomber,bool * gposbitmap,float *gdictionary)
{

	int streamnumber=10;
	hipMemcpy(gqts, qts,sizeof(float)*16,hipMemcpyHostToDevice);
	unsigned long int datasize=100000000;

	int aaa=startnumber/100000000;
	int bbb=stopnomber/100000000+1;

	if(bbb>=10)
	bbb=10;


	hipStream_t streams[streamnumber];
	for(int i=0;i<streamnumber;i++)
	{
		hipStreamCreate(&streams[i]);
	}
	for(int i=0;i<streamnumber;i++)
	{
		calculate_lbd3<<<200,500,10,streams[i]>>> (saxarray+i*datasize*16/streamnumber,gqts, datasize/streamnumber, 16,gposbitmap+i*datasize/streamnumber,BSF,16.0); 
       	hipMemcpyAsync(posbitmap+i*datasize/streamnumber, gposbitmap+i*datasize/streamnumber, sizeof(bool)*datasize/streamnumber,hipMemcpyDeviceToHost,streams[i]);
	}

	hipDeviceSynchronize();

}
extern "C" void SIMSlowerGPUgrid(sax_type **saxarray, bool **posbitmap,float * qts,float * gqts,float BSF,long unsigned datasize,bool ** gposbitmap,float *gdictionary,long int *nodemap, unsigned long *gridnumber)
{
	hipMemcpy(gqts, qts,sizeof(float)*16,hipMemcpyHostToDevice);
	for(long int i=0;i<datasize;i++)
	{

		calculate_lbd2<<<500,200,10>>> (saxarray[nodemap[i]],gqts, gridnumber[nodemap[i]], 16,gdictionary, gposbitmap[nodemap[i]],BSF); 
	}
	for(long int i=0;i<datasize;i++)
	{
		hipMemcpy(posbitmap[nodemap[i]], gposbitmap[nodemap[i]], sizeof(bool)*gridnumber[nodemap[i]],hipMemcpyDeviceToHost);
	}
	hipDeviceSynchronize();
}

__global__ void calculate_lbd4(const sax_type * const saxarray,const float * const paa, const long int M, const int N,bool * positionarray,const float BSF,const long int offset) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float distance = 0;
	float lbsf=BSF/16.0;

	int i=0;
        		float breakpoint_lower = 0; // <-- TODO: calculate breakpoints.
        		float breakpoint_upper = 0; // <-- - || -

	
	for (int j = thid; j < M; j += gridDim.x*blockDim.x)
	{
		distance=0;
		for (i=0; i<N; i++) {
                	if(distance<lbsf)
		{
        	
        		sax_type v = saxarray[j*N+i];

        		sax_type region_lower = v ;//shift operation 
        		sax_type region_upper = (~((int)MAXFLOAT) | region_lower);



        	
        		if (region_lower == 0)
			{
	            		breakpoint_lower = -2000000;
				float breaku=((float)region_lower-127.0f)/128.0f;
            			breakpoint_upper = breaku*(1.1362582192*breaku*breaku+0.99800);//sax_breakpoints[region_upper+1];//search in a list(why?)
				if(breakpoint_upper < paa[i])
				{
            				distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
        			}
			}
        		else if (region_upper == 256 - 1) 
			{
            			breakpoint_upper = +2000000;
				float breakx=((float)region_lower-128.0f)/128.0f;
           			breakpoint_lower = breakx*(breakx*breakx*1.1362582192+0.99800);//sax_breakpoints[region_lower];
				if (breakpoint_lower > paa[i]) 
				{
            				distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
        			}
        		}
        		else 
			{
				float breakx=((float)region_lower-128.0f)/128.0f;
           			breakpoint_lower = breakx*(breakx*breakx*1.1362582192+0.99800);//sax_breakpoints[region_lower];
				if (breakpoint_lower > paa[i]) 
				{
            				distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
        			}
				else
				{
					float breaku=((float)region_lower-127.0f)/128.0f;
            				breakpoint_upper = breaku*(1.1362582192*breaku*breaku+0.99800);//sax_breakpoints[region_upper+1];//search in a list(why?)
					if(breakpoint_upper < paa[i])
					{
            					distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
        				}
        			} 

        		}
		}
						        		
    	}

		if(distance<lbsf)
		{positionarray[j]=true;}
		else
		{positionarray[j]=false;}
	}
}


__global__ void calculate_lbd2float(const sax_type * const saxarray,const float * const paa, const long int M, const int N,float * const sax_breakpoints,float * positionarray,const float BSF) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float distance = 0;
	

	int i=0;
        		float breakpoint_lower = 0; // <-- TODO: calculate breakpoints.
        		float breakpoint_upper = 0; // <-- - || -


	for (int j = thid; j < M; j += gridDim.x*blockDim.x)
	{
		distance=0;
		for (i=0; i<N; i++) 
		{
        	if(16*distance<BSF)
		{
        	
        		sax_type v = saxarray[j*N+i];

        		sax_type region_lower = v ;//shift operation 
        		sax_type region_upper = (~((int)MAXFLOAT) | region_lower);



        	
        		if (region_lower == 0)
			{
	            		breakpoint_lower = -2000000;
        		}
        		else
        		{
				float breakx=((float)region_lower-128.0f)/128.0f;
           			breakpoint_lower = breakx*(breakx*breakx*1.1362582192+0.99800);//sax_breakpoints[region_lower];
        		}




        		if (region_upper == 256 - 1) 
			{
            			breakpoint_upper = +2000000;
        		}
        		else 
			{
			float breaku=((float)region_lower-127.0f)/128.0f;
            			breakpoint_upper = breaku*(1.1362582192*breaku*breaku+0.99800);//sax_breakpoints[region_upper+1];//search in a list(why?)
        		} 

	        			

        		if (breakpoint_lower > paa[i]) 
			{

            			distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
        		}
        		else if(breakpoint_upper < paa[i])
			{
            			distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
        		}
		}
			        				        		
    		}


		positionarray[j]=16*distance;
	}
}

__global__ void calculate_ed2(const float * const Subject, const float * const Query, const long int M, const int N,float * gposbitmap) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float dist=0;
	long int poss;
	//printf("hello jfeowjfw %d\n",thid);
	for (int i = thid; i < M; i += gridDim.x*blockDim.x)
	{	
		dist=0;
		poss=i*N;
		for(int j =0;j<N;j++)
		{
        		dist += (Subject[poss+j]-Query[j])*(Subject[poss+j]-Query[j]);
				
		}
		gposbitmap[i]=dist;
		//if(dist<BSF)
		//resultmap[i]=dist;
	}		
}
__global__ void calculate_lbdold(const sax_type * const saxarray,const float * const paa, const long int M, const int N,float * const sax_breakpoints,bool * positionarray,const float BSF) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float distance = 0;
	

	int i=0;
        		float breakpoint_lower = 0; // <-- TODO: calculate breakpoints.
        		float breakpoint_upper = 0; // <-- - || -


	for (int j = thid; j < M; j += gridDim.x*blockDim.x)
	{
		distance=0;
		for (i=0; i<N; i++) {
        
        	
        		sax_type v = saxarray[j*N+i];

        		sax_type region_lower = v ;//shift operation 
        		sax_type region_upper = (~((int)MAXFLOAT) | region_lower);



        	
        		if (region_lower == 0)
			{
	            		breakpoint_lower = -2000000;
        		}
        		else
        		{
           			breakpoint_lower = sax_breakpoints[region_lower];//(float)(region_lower-128)*(region_lower-128)/16484.0f;//sax_breakpoints[region_lower];
        		}

        		if (region_upper == 256 - 1) 
			{
            			breakpoint_upper = +2000000;
        		}
        		else
        		{
            			breakpoint_upper = sax_breakpoints[region_lower+1];//(float)(region_upper+1-128)*(region_upper+1-128)/16484.0f;//sax_breakpoints[region_upper+1];//search in a list(why?)
        		}

	        			

        		if (breakpoint_lower > paa[i]) 
			{

            			distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
        		}
        		else if(breakpoint_upper < paa[i])
			{
            			distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
        		}

    		}

		if(16*distance<BSF)
		{positionarray[j]=true;}
		else
		{positionarray[j]=false;}
	}
}
__global__ void calculate_lbd2(const sax_type * const saxarray,const float * const paa, const long int M, const int N,float * const sax_breakpoints,bool * positionarray,const float BSF) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float distance = 0;
	

	int i=0;
        		float breakpoint_lower = 0; // <-- TODO: calculate breakpoints.
        		float breakpoint_upper = 0; // <-- - || -


	for (int j = thid; j < M; j += gridDim.x*blockDim.x)
	{
		distance=0;
		for (i=0; i<N; i++) {
                	if(16*distance<BSF)
		{
        	
        		sax_type v = saxarray[j*N+i];

        		sax_type region_lower = v ;//shift operation 
        		sax_type region_upper = (~((int)MAXFLOAT) | region_lower);



        	
        		if (region_lower == 0)
			{
	            		breakpoint_lower = -2000000;
        		}
        		else
        		{
				float breakx=((float)region_lower-128.0f)/128.0f;
           			breakpoint_lower = breakx*(breakx*breakx*1.1362582192+0.99800);//sax_breakpoints[region_lower];
        		}

        		if (region_upper == 256 - 1) 
			{
            			breakpoint_upper = +2000000;
        		}
        		else 
			{
			float breaku=((float)region_lower-127.0f)/128.0f;
            			breakpoint_upper = breaku*(1.1362582192*breaku*breaku+0.99800);//sax_breakpoints[region_upper+1];//search in a list(why?)
        		} 

	        			

        		if (breakpoint_lower > paa[i]) 
			{

            			distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
        		}
        		else if(breakpoint_upper < paa[i])
			{
            			distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
        		}
}
						        		
    		}

		if(16*distance<BSF)
		{positionarray[j]=true;}
		else
		{positionarray[j]=false;}
	}
}
extern "C" void SIMSlowerGPUsmall(sax_type *saxarray, bool *posbitmap,float * qts,float * gqts,float BSF,long unsigned datasize,bool * gposbitmap,float *gdictionary)
{

	calculate_lbd<<<20,20,10>>> (saxarray,gqts, datasize, 16,gdictionary, gposbitmap,BSF); 

	hipMemcpy(posbitmap, gposbitmap, sizeof(bool)*datasize,hipMemcpyDeviceToHost);

}
extern "C" void copyqts(float * qts,float * gqts)
{
	hipMemcpy(gqts, qts,sizeof(float)*16,hipMemcpyHostToDevice);
}
*/